#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
//#include <hip/hip_runtime.h> 

#include "cuda_wrapper.h"

__global__ void print_from_gpu(void)
{
	printf("Device:Hello World! from thread [%d, %d]\
		From device\n", threadIdx.x, blockIdx.x);	
}

namespace Wrapper
{
	void wrapper(void)
	{
		printf("Host: Hello, world!\n"); 
		print_from_gpu<<<2,1>>>(); 
		hipDeviceSynchronize();
	}
}
